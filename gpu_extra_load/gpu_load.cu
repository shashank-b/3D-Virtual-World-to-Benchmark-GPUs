
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <ctime>
#include <random>
#include <iostream>
#include <cmath>
#include <stdio.h>
#include <iomanip>
#include <chrono>
#include <unistd.h>
#include <map>

using namespace std::chrono;
using namespace std;
#define N_THREADS_PER_BLOCK 1024

__device__ double atomicAddDouble(double* address, double val)
{
    unsigned long long int* address_as_ull =
                             (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val +         __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}

class MatrixCUDA{
    public:
        int *dim;
        float *matrix;
        int *dev_dim;
        float *dev_matrix;

        MatrixCUDA(int *n){
            
            matrix = (float* )malloc(sizeof(float)*(*n)*(*n));
            
            hipMalloc( (void**)&dev_matrix, sizeof(float)*(*n)*(*n));
            for(int i=0; i< (*n)*(*n); i++){
        
        		matrix[i] = 0.0;
   			}

    		hipMemcpy( dev_matrix, matrix, sizeof(float)*(*n)*(*n), hipMemcpyHostToDevice );
                  
            dim = (int* )malloc(sizeof(int));
            *dim = *n;
            hipMalloc((void**)&dev_dim, sizeof(int) );
            hipMemcpy( dev_dim, dim, sizeof(int), hipMemcpyHostToDevice);
            
        }

        void random_init(float *lower_limit, float *higher_limit);
        void print_matrix();
        void clear_mem();
};

void MatrixCUDA::clear_mem(){

    /*for(int i=0 ; i<(*dim)*(*dim); i++) {
        cudaFree(dev_matrix[i]);
    
    }*/
    free(dim);
    hipFree(dev_dim);
    
    free(matrix);
    hipFree(dev_matrix);

    return;

}

void MatrixCUDA::random_init(float *lower_limit, float *higher_limit){
    //float **random_numbers;
        //random_numbers = (float** )malloc(sizeof(float*)*(*dim));
    
            
    for(int i=0; i< (*dim)*(*dim); i++){
        //random_numbers[i] = (float* )malloc(sizeof(float)*(*dim));
        //for(int j=0; j< *dim; j++){
            //float random_number;
            //random_number =  (float* )malloc(sizeof(float));
            
            //random_number = 
            matrix[i] = *lower_limit + static_cast <float> (rand()) /(static_cast <float> (RAND_MAX/( *higher_limit - *lower_limit)));
            //free(random_number);

        //}

    }

    hipMemcpy( dev_matrix, matrix, sizeof(float)*(*dim)*(*dim), hipMemcpyHostToDevice );

    return;
}

void MatrixCUDA::print_matrix(){
    

    hipMemcpy( matrix, dev_matrix, sizeof(float)*(*dim)*(*dim), hipMemcpyDeviceToHost );   
   
    for(int i=0; i<*dim; i++){
        for(int j=0; j<*dim; j++){
            cout << matrix[i*(*dim)+j]<< "  ";
        }
        cout << endl;
    }


    cout << "Dimensions is " << *dim << " X " << *dim << endl;

    return;
}

    
__global__ void mult(float *a_matrix, float *b_matrix, float *c_matrix, int *dim){
    
    int n = *dim;
    __shared__ float temp_sum[N_THREADS_PER_BLOCK];

    int z = int((n+N_THREADS_PER_BLOCK-1)/N_THREADS_PER_BLOCK);
    int dim1 = int((blockIdx.x)/(z*n));
    int dim2 = int((blockIdx.x)/z)%n;
    int dim3 = (blockIdx.x)%z;

	int index = dim3*(N_THREADS_PER_BLOCK) + threadIdx.x;
	
	if( index > (*dim)){
    	temp_sum[threadIdx.x] = 0;
    }

    else{
	temp_sum[threadIdx.x] = a_matrix[dim1*(*dim) + index]* b_matrix[(index)*(*dim) + dim2];
	// (*counta)++;
	}
    __syncthreads();

    if( 0 == threadIdx.x){
        float block_sum = 0;
        for (int i = 0; i < (N_THREADS_PER_BLOCK) ; i++){
            block_sum += temp_sum[i];
            // (*countb)++;	
        }
        float new_sum = block_sum;
        atomicAdd(&(c_matrix[dim1*(*dim)+dim2]), new_sum);
    }
    return;
}

int main(){
    //   500 - 51
    //  1000 - 51
    //  2000 - 86
    //  4000 - 223
    //  8000 - 753
    // 10000 - 1165
    // 12000 - 1669
    // 14000 - 2264
    // 16000 - 1991
    // 18000 - 2493
    // 20000 - 1547
    // 22000 - 1867
    // 26000 - 2598

	std::map<int, int> load;
	
	load[50] = 1000;
	load[75] = 2000;
	load[200] = 4000;
	load[750] = 8000;
	load[1100] = 10000;
	load[1600] = 12000;
	load[1850] = 22000;
	load[2000] = 16000;
	load[2200] = 14000;
	load[2500] = 18000;
	load[2600] = 26000;
	int load_wanted;// = 8000;
    cout << "Enter load in MB and press enter" << endl;
    cout << "Select from the following (50, 75, 200, 750, 1100, 1600, 1850, 2000, 2200, 2500 or 2600)" << endl;

    srand (static_cast <unsigned> (5));
    cin >> load_wanted;
    cout << "Loaded" << endl;
    cout << "Press Ctrl C if you love your GPU" << endl;
    int matrix_dim = load[load_wanted];
    while(true){
        MatrixCUDA a(&matrix_dim);
        MatrixCUDA b(&matrix_dim);
        MatrixCUDA c(&matrix_dim);
        float lower_limit;
        float upper_limit;

        lower_limit = -2;
        upper_limit = 2;
        a.random_init(&lower_limit, &upper_limit);

        b.random_init(&lower_limit, &upper_limit);
       

        
        int dimension3 =  ceil((matrix_dim*1.0)/float(N_THREADS_PER_BLOCK));
        int n_blocks = (matrix_dim*matrix_dim*dimension3);
         float dur;
        hipEvent_t start, stop;

        hipEventCreate(&start) ;
        hipEventCreate(&stop) ;
        hipEventRecord(start, 0) ;

        
        mult <<< n_blocks, N_THREADS_PER_BLOCK, N_THREADS_PER_BLOCK>>>(a.dev_matrix, b.dev_matrix, c.dev_matrix, a.dev_dim);

        hipEventRecord(stop, 0) ;
        hipEventSynchronize(stop) ;
        hipEventElapsedTime(&dur, start, stop);

        a.clear_mem();
        b.clear_mem();
        c.clear_mem();                
    }
    //}
    
    return 0;
}


    // srand (static_cast <unsigned> (5));
    // int matrix_size[4] = {16, 128, 1024,1024*16,};
    // //cout << "Matrix dimension" << "     N_THREADS_PER_BLOCK" << "    time(microseconds)"<< endl;
    // //for (int i =0 ; i<4; i++){
    //     int matrix_dim = 1024;// matrix_size[i];
    //     MatrixCUDA a(&matrix_dim);
    //     MatrixCUDA b(&matrix_dim);
        
    //     MatrixCUDA c(&matrix_dim);
    //     float lower_limit;
    //     float upper_limit;

    //     lower_limit = -1000.0;
    //     upper_limit = 1000.0;
    //     a.random_init(&lower_limit, &upper_limit);
        
    //     //a.print_matrix();
    //     b.random_init(&lower_limit, &upper_limit);
    //     c.random_init(&lower_limit, &upper_limit);

    //     //c.print_matrix();
    //     //int threads[9] = {4,8,16,32,64,128,256,512,1024};

    //     //for(int j = 0; j<9; j++){
    //         int average = 0;
            
    //         const int N_THREADS_PER_BLOCK = 1024;//threads[j];///threads[j];

    //         // if (N_THREADS_PER_BLOCK > matrix_dim){
    //         //     break;
    //         // }  
    //         //for (int k = 0; k<10; k++){
                
    //             //cudaEvent_t start, stop;
    //             //cudaEventCreate(&start);
    //             //cudaEventCreate(&stop);
    //             //cudaEventRecord(start);
    //             //cout<<"here"<<endl;
    //             auto start = high_resolution_clock::now(); 
    //             int n_blocks = ceil((matrix_dim*matrix_dim*matrix_dim*1.0)/float(N_THREADS_PER_BLOCK));
    //             cout<<n_blocks<<" "<<N_THREADS_PER_BLOCK << endl;
    //             mult <<< n_blocks, N_THREADS_PER_BLOCK, N_THREADS_PER_BLOCK>>>(a.dev_matrix, b.dev_matrix, c.dev_matrix, a.dev_dim, N_THREADS_PER_BLOCK);
    //             //cout<<"2"<<endl;
    //             //cudaEventRecord(stop);
    //             //cout << "end"<<endl;
    //             //cudaEventSynchronize(stop);
    //             //float milliseconds = 0;
    //             //cudaEventElapsedTime(&milliseconds, start, stop);
                
    //             auto stop = high_resolution_clock::now();
                
    //             auto duration = duration_cast<nanoseconds>(stop - start); 
    //             //cout << "Matrix dimension is " << matrix_dim << " X "<<  matrix_dim << " and the N_THREADS_PER_BLOCK are " << N_THREADS_PER_BLOCK << "and the time take is " << duration.count() << " microseconds" << endl;//<< milliseconds << endl;//
    //         //     if(k >1){
    //         //         average += duration.count();
    //         //     }
    //         // }

    //         average =average/8;

    //         cout << matrix_dim << " X "<<  matrix_dim << " \t \t " << N_THREADS_PER_BLOCK << " \t \t " << average << endl;
             
    //     //}
    //     // a.print_matrix();
    //     // b.print_matrix();
    //     //c.print_matrix();
    //     a.clear_mem();
    //     b.clear_mem();dim
    //     c.clear_mem();                
    
    // //}
